#include "hip/hip_runtime.h"
#include <glad/glad.h>
#include <GLFW/glfw3.h>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <stdio.h>
#include <iostream>

#include "renderer/render.h"

#define errCheck(call)                                                      \
{                                                                           \
    const hipError_t error = call;                                         \
    if (error != hipSuccess) {                                             \
        fprintf(stderr, "Error: %s:%d\n", __FILE__, __LINE__);              \
        fprintf(stderr, "code: %d, reason: %s\n",                           \
                error, hipGetErrorString(error));                          \
    }                                                                       \
}

void print_gpu_prop();
void first_method();

int main() {
    print_gpu_prop();
    first_method();
}

void print_gpu_prop() {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    printf("Number of devices: %d\n", nDevices);

    for (int device=0; device<nDevices; device++) {
        hipDeviceProp_t deviceProp;

        hipGetDeviceProperties(&deviceProp, device);
        hipDeviceSynchronize();

        printf("Device name: %s\n", deviceProp.name);
        printf("  Compute capability:                   %d.%d\n",
                    deviceProp.major, deviceProp.minor);
        printf("  Concurrent kernels:                   %s\n",
                    deviceProp.concurrentKernels ? "yes" : "no");
        printf("  Concurrent computation/communication: %s\n",
                    deviceProp.deviceOverlap ? "yes" : "no");
        printf("  Clock Rate (kHz):                     %d\n",
                    deviceProp.clockRate);
        printf("  Memory Clock Rate (MHz):              %d\n",
                    deviceProp.memoryClockRate/1000);
        printf("  Memory Bus Width (bits):              %d\n",
                    deviceProp.memoryBusWidth);
        printf("  Peak Memory Bandwidth (GB/s):         %.1f\n",
                    2.0*deviceProp.memoryClockRate*((float)deviceProp.memoryBusWidth/8)/1.0e6);
        printf("  ---\n");

        printf("  Total global memory (Gbytes)          %.1f\n",
                    (float)(deviceProp.totalGlobalMem)/1024.0/1024.0/1024.0);
        printf("  Total constant memory (Kbytes)        %.1f\n",
                    (float)(deviceProp.totalConstMem)/1024.0);
        printf("  Shared memory per block (Kbytes)      %.1f\n",
                    (float)(deviceProp.sharedMemPerBlock)/1024.0);
        printf("  Shared memory per MP (Kbytes)         %.1f\n",
                    (float)(deviceProp.sharedMemPerMultiprocessor)/1024.0);
        printf("  ---\n");

        printf("  Warp-size:                            %d\n",
                    deviceProp.warpSize);
        printf("  Max threads per block:                %d\n",
                    deviceProp.maxThreadsPerBlock);
        printf("  Max threads dim:                      (%d,%d,%d)\n",
                    deviceProp.maxThreadsDim[0], deviceProp.maxThreadsDim[1], deviceProp.maxThreadsDim[2]);
        printf("  Max grid size:                        (%d,%d,%d)\n",
                    deviceProp.maxGridSize[0], deviceProp.maxGridSize[1], deviceProp.maxGridSize[2]);
        printf("  ---\n");

        printf("  Number of multiprocessors:            %d\n",
                    deviceProp.multiProcessorCount);
        printf("  Max blocks per multiprocessors:       %d\n",
                    deviceProp.maxBlocksPerMultiProcessor);
        printf("  Max threads per multiprocessors:      %d\n",
                    deviceProp.maxThreadsPerMultiProcessor);
        printf("  Max regs per multiprocessors:         %d\n",
                    deviceProp.regsPerMultiprocessor);

    }
    hipDeviceReset();
}

__global__ void
updateTexture(unsigned char* d_textureData, int width, int height, int i) {
    const int x = threadIdx.x + blockIdx.x * blockDim.x;
    const int y = threadIdx.y + blockIdx.y * blockDim.y;

    if ((x<width) & (y<height)) {
        int idx = y * width + x;
        d_textureData[idx * 4 + 0] = (255 + x*i)%255;
        d_textureData[idx * 4 + 1] = (255 + y*i)%255;
        d_textureData[idx * 4 + 2] = (255 + (x+y)*i)%255;
        d_textureData[idx * 4 + 3] = 255;
    }
}

void first_method(){
    // --------------------------
    // OpenGL setup
    Render::setup_opengl(3, 3);
    GLFWwindow* window = Render::create_window(1024, 768, "Cuda_OpenGL_Interop");
    Render::setup_glad();

    GLuint shaderProgram;
    Render::create_shader_program(&shaderProgram);

    GLuint quad_vao;
    Render::create_quad(&quad_vao);

    int width = 2;
    int height = 2;

    GLuint texture;
    glGenTextures(1, &texture);
    glBindTexture(GL_TEXTURE_2D, texture);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, width, height, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr); // Initialize texture
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glBindTexture(GL_TEXTURE_2D, 0);

    // Check which GPU is used
    const GLubyte* vendor = glGetString(GL_VENDOR); // Returns the vendor
    const GLubyte* renderer = glGetString(GL_RENDERER); 
    std::cout << vendor << std::endl;
    std::cout << renderer << std::endl;

    // Register the texture with CUDA
    struct hipGraphicsResource* cudaResource;
    errCheck(hipGraphicsGLRegisterImage(&cudaResource, texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsWriteDiscard));

    // Update texture
    hipArray* cuArray;
    unsigned char* d_textureData;
    errCheck(hipMalloc(&d_textureData, width*height*sizeof(unsigned char)*4));
    errCheck(hipDeviceSynchronize());
    dim3 blockDim(16, 16, 1);
    dim3 gridDim( (width+blockDim.x-1)/blockDim.x, (height+blockDim.y-1)/blockDim.y, 1 );

    glfwSwapInterval(1);
    int i = 0;
    while (!glfwWindowShouldClose(window)) {
        glClearColor(0.3, 0.5, 0.7, 1);
        glClear(GL_COLOR_BUFFER_BIT);

        errCheck(hipGraphicsMapResources(1, &cudaResource, 0));
        errCheck(hipGraphicsSubResourceGetMappedArray(&cuArray, cudaResource, 0, 0));            // Get a hipArray to actually be able to access texture data
        updateTexture<<<gridDim, blockDim>>>(d_textureData, width, height, i++);
        errCheck(hipMemcpy2DToArray(cuArray, 0, 0, d_textureData, width*sizeof(unsigned char)*4, width*sizeof(unsigned char)*4, height, hipMemcpyDefault));
        errCheck(hipGraphicsUnmapResources(1, &cudaResource, 0));
        errCheck(hipDeviceSynchronize());

        Render::draw_quad(shaderProgram, quad_vao, texture);

        glfwSwapBuffers(window);
        glfwPollEvents();
    }

    hipFree(d_textureData);
    glDeleteTextures(1, &texture);
    glfwDestroyWindow(window);
    glfwTerminate();
}
